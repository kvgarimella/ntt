
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// iterates over list of floats x and y
// adds x + y -> y
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

__global__
void add_gpu(int n, float *x, float *y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx < n)
    {
        y[idx] = x[idx] + y[idx];
    }
}

int main(void)
{
    int N = 32; // basically 1mil 
    printf("Number of elements: %d\n", N);

    float *x;
    float *y;
    x = (float *) malloc(N * sizeof(float));
    y = (float *) malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    float *x_gpu, *y_gpu;
    hipMalloc((void **) &x_gpu, N * sizeof(float));
    hipMalloc((void **) &y_gpu, N * sizeof(float));
    hipMemcpy(x_gpu, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, N*sizeof(float), hipMemcpyHostToDevice);



    // numbers in triple brackets will parameterize GPU usage for function call
    // lets change <<<1,1>>> to <<<1,256>>> i.e. increase thread to 256
    // must be a multiple of 32
    add_gpu<<<2,16>>>(N, x_gpu, y_gpu);
    //add(N, x, y);

    hipMemcpy(y, y_gpu, N*sizeof(float), hipMemcpyDeviceToHost);
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));

    printf("Max error: %f\n", maxError);
    for (int i = 0; i < N; i++)
        printf("%f, ", y[i]);
    printf("\n");

    hipFree(x_gpu);
    hipFree(y_gpu);
    free(x);
    free(y);

    return 0;
}
