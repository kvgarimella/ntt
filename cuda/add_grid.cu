
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// iterates over list of floats x and y
// adds x + y -> y
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

__global__
void add_gpu(int n, float *x, float *y)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; // basically 1mil 
    printf("Number of elements: %d\n", N);

    float *x;
    float *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    // numbers in triple brackets will parameterize GPU usage for function call
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add_gpu<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));

    printf("Max error: %f\n", maxError);

    hipFree(x);
    hipFree(y);

    return 0;
}
