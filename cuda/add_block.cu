
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// iterates over list of floats x and y
// adds x + y -> y
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

__global__
void add_gpu(int n, float *x, float *y)
{
    int index  = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i+= stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; // basically 1mil 
    printf("Number of elements: %d\n", N);

    float *x;
    float *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    // numbers in triple brackets will parameterize GPU usage for function call
    // lets change <<<1,1>>> to <<<1,256>>> i.e. increase thread to 256
    // must be a multiple of 32
    add_gpu<<<1,256>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));

    printf("Max error: %f\n", maxError);

    hipFree(x);
    hipFree(y);

    return 0;
}
