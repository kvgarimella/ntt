
#include <hip/hip_runtime.h>
__device__ uint64_t modulo(int64_t base, int64_t m){
	int64_t result = base % m;

	return (result >= 0) ? result : result + m;
}

uint64_t modulo_cpu(int64_t base, int64_t m){
	int64_t result = base % m;

	return (result >= 0) ? result : result + m;
}

__device__ uint64_t modExp(uint64_t base, uint64_t exp, uint64_t m){

	uint64_t result = 1;
	
	while(exp > 0){

		if(exp % 2){

			result = modulo(result*base, m);

		}

		exp = exp >> 1;
		base = modulo(base*base,m);
	}

	return result;
}
uint64_t modExp_cpu(uint64_t base, uint64_t exp, uint64_t m){

	uint64_t result = 1;
	
	while(exp > 0){

		if(exp % 2){

			result = modulo_cpu(result*base, m);

		}

		exp = exp >> 1;
		base = modulo_cpu(base*base,m);
	}

	return result;
}


__global__ void inner_loop(uint64_t *result, uint64_t n, uint64_t p, uint64_t m, uint64_t a, uint64_t batch){
    uint64_t factor1, factor2;

    int idx = threadIdx.x;
    int j = m*blockIdx.x;
    int k = blockIdx.y;
    if ((j + k + m/2 + idx*n) < n*batch){ 

				factor1 = result[j + k + idx * n];
				factor2 = modulo(modExp(a,k,p)*result[j + k + m/2 + idx * n],p);

			
				result[j + k + idx * n] 		= modulo(factor1 + factor2, p);
				result[j + k+m/2 + idx * n] 	= modulo(factor1 - factor2, p);
   } 

}

__host__ void inPlaceNTT_DIT(uint64_t *result, uint64_t n, uint64_t p, uint64_t r, bool rev, uint64_t batch){

	uint64_t m, k_, a;
	for(uint64_t i = 1; i <= log2(n); i++){ 

		m = pow(2,i);
		k_ = (p - 1)/m;
		a = modExp_cpu(r,k_,p);
        dim3 blocks(n/m, m/2, 1);
        dim3 threads(batch, 1, 1);
        inner_loop<<<blocks, threads>>>(result,n,p,m,a,batch); 
	}
}


