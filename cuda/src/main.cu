#include "hip/hip_runtime.h"
#include <cmath>		/* pow() */
#include <cstdint>		/* uint64_t */
#include <ctime>		/* time() */
#include <cstdlib>

#include <unistd.h>
#include <iostream>

#include "ntt.cu"
#include "utils.cu"


using namespace std;
#include <ctime>		/* time() */
#include <sys/time.h>
#include <stdlib.h>
#include <iostream>
#include <cstdint> 		

int main(int argc, char *argv[]){
  if (argc < 3)
  {
      printf("Usage: ./ntt [Vector Size] [Batch Size]\n");
      printf("You must enter a size for the vector and the number of vectors!\n");
      printf("Please make the vector a power of 2\n");
      return -1;
  }

  uint64_t n     = atoi(argv[1]);
  uint64_t batch = atoi(argv[2]);


  uint64_t p = 1073750017;
  uint64_t r = 5;
  bool t     = true;

  int size = n * batch*sizeof(uint64_t);
  uint64_t *vec;
  vec = (uint64_t *) malloc(size);
  for (int kk = 0; kk < batch; ++kk){
      for (int ii = 0; ii < n; ++ii)
          vec[kk*n + ii] = ii;
  }
  printf("Original vector: ");
  printVec(vec, n);

  uint64_t *result_gpu, *vec_gpu, *result_cpu;
  result_cpu = (uint64_t *) malloc(size);

  hipMalloc((void**)&vec_gpu, size);
  hipMemcpy(vec_gpu, vec, size, hipMemcpyHostToDevice);


  result_gpu = bit_reverse_table(vec_gpu, n, batch);
  inPlaceNTT_DIT(result_gpu, n, p, r, t, batch); 


  hipMemcpy(result_cpu, result_gpu, size, hipMemcpyDeviceToHost);
  printf("Final vector   : ");
  printVec(result_cpu, n);

  hipFree(result_gpu);
  hipFree(vec_gpu);
  free(result_cpu);
  free(vec);

  return 0;

}

